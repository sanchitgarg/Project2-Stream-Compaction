#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <cmath>
#include <iostream>

namespace StreamCompaction {
namespace Naive {

#define SHOW_TIMING 0
int numBlocks, numThreads = 256;

int * dev_odata;
int * dev_idata;

 __global__ void scanStep(int n, int jump, int *odata, int *idata)
 {
	 int index = threadIdx.x + (blockIdx.x * blockDim.x);

	 if(index >= jump && index < n)
	 {
		 odata[index] = idata[index] + idata[index - jump];
	 }
 }

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {

    hipMalloc((void**)&dev_odata, n * sizeof(int));
    hipMalloc((void**)&dev_idata, n * sizeof(int));

    hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
	numBlocks = n / numThreads + 1;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int p=1;
    for(int i=1; i< ilog2ceil(n)+1; ++i)
    {
    	scanStep<<<numBlocks, numThreads>>>(n, p, dev_odata, dev_idata);
    	p <<= 1;
    	hipMemcpy(dev_idata, dev_odata, n * sizeof(int), hipMemcpyDeviceToDevice);
    }

    hipEventRecord(stop);
   	hipEventSynchronize(stop);
   	float milliseconds = 0;
   	hipEventElapsedTime(&milliseconds, start, stop);
   	if(SHOW_TIMING)
   	   	std::cout<<"Total time in milliseconds : "<<milliseconds<<std::endl;

    hipMemcpy(odata+1, dev_odata, (n-1) * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_odata);
    hipFree(dev_idata);
}

}
}
